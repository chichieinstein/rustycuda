#include "../include/device_C_interface.cuh"

extern "C"
{
    void* memory_allocate_device(int size)
    {
        void* inter;
        hipMalloc((void**)&inter, size);
        return inter;
    }

    void memory_deallocate_device(void* inter)
    {
        hipFree(inter);
    }

    void transfer(void* cpu_arr, void* gpu_arr, int count)
    {
        hipMemcpy(cpu_arr, gpu_arr, count, hipMemcpyDeviceToHost);
    }
}