#include "../include/device_C_interface.cuh"
using std::cyl_bessel_if;

extern "C"
{
    void* memory_allocate_device(int size)
    {
        void* inter;
        hipMalloc((void**)&inter, size);
        return inter;
    }

    void memory_deallocate_device(void* inter)
    {
        hipFree(inter);
    }

    void transfer(void* cpu_arr, void* gpu_arr, int count)
    {
        hipMemcpy(cpu_arr, gpu_arr, count, hipMemcpyDeviceToHost);
    }

    float bessel_func(float inp)
    {
        return cyl_bessel_if(0.0, inp);
    }
}